
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <cmath>

__device__ int position;			//index of the largest value
__device__ int largest;				//value of the largest value
int lenString = 593;
int maxNumStrings = 1000000;                           
int threshold = 2;

__global__ void anyLeft(int *d_c, int *remaining, int max_count) {
	int my_id = blockDim.x * blockIdx.x + threadIdx.x;
	if((d_c[my_id] == 0) && (my_id < max_count)) {
		*remaining = 0;
	}
}

__global__ void search(int *d_b, int *d_c, int max_count) {
	int my_id = blockDim.x * blockIdx.x + threadIdx.x;
	if((d_c[my_id] == 0) && (d_b[my_id] == largest) && (my_id < max_count)) {
		position = my_id;
	}
}

__global__ void populate (int *d_b, int *copy_db, int *d_c, int size, int *left) {
	int n = 0;
	*left = 1;

	int my_id = blockDim.x * blockIdx.x + threadIdx.x;

	if (my_id < size) {
		n = abs((bool)d_c[my_id] - 1);
		copy_db[my_id] = d_b[my_id] * n;
	}		
}

__device__ void cuda_select(int *db, int size) {
	int my_id = blockDim.x * blockIdx.x + threadIdx.x;

	if(my_id < size) {
		if(db[2 * my_id] > db[2 * my_id + 1])
			db[my_id] = db[2 * my_id];
		else
			db[my_id] = db[2 * my_id + 1];
	}	
}

__global__ void select(int *db, int size) {
	int height = (int)ceil(log2((double)size));
	int i = 0;
	
	for(i = 0; i < height; i++) {
		size = (int)ceil((double) size/2);
		cuda_select(db, size);
	}
	largest = db[0];
}

__global__ void compare(char *d_a, int *d_b, int *d_c, int max_count, int lenString, int threshold) {

	int my_id = blockDim.x * blockIdx.x + threadIdx.x;

	if (my_id == position) 
		d_c[my_id] = 2;
	
		
	if ((my_id < max_count) && (d_c[my_id] == 0) && (my_id != position)) {	
		int x, diffs = 0;

		for (x = 0; x < lenString; x++) {
			diffs += (bool)(d_a[(lenString*position)+x]^d_a[(my_id*lenString)+x]);
			
			if (diffs > threshold)
				break;
		}
		
		if (diffs <= threshold) {
			d_b[position] += d_b[my_id];
			d_c[my_id] = 1;
		}
	} 
}

int main(int argc, char** argv) {
	char *strings, *d_a;
	int *counts, *merged, *d_b, *d_c;
	int *largest, *copy_db;
	char copy[lenString+1];
	int numbers = 0;
	int i = 0, actual_count = 0;
	int size_string = maxNumStrings*sizeof(char)*(lenString+1);
	int size_int = maxNumStrings*sizeof(int);
	
	int *any_left, *left;

	//opening the file
	FILE *fp;
	fp = fopen("/cluster/home/charliep/courses/cs360/single-linkage-clustering/Iceland2014.trim.contigs.good.unique.good.filter.unique.count.fasta", "r");

	if (!(strings= (char *)malloc(size_string))) {
		fprintf(stderr, "malloc() FAILED (Block)\n"); 
		exit(0);
	}

	if (!(counts= (int*)malloc(size_int))) {
		fprintf(stderr, "malloc() FAILED (Block)\n"); 
		exit(0);
	}

	merged = (int *)malloc(size_int);
	any_left = (int *)malloc(sizeof(int));
	hipMemset(&position, 0, sizeof(int));
	hipMemset(&largest, 0, sizeof(int));

	while(fscanf(fp, "%s %d", copy, &numbers) != EOF && actual_count <1000){
		strcpy(&strings[i], copy);
		counts[actual_count] = numbers;
		
		i = i + lenString;
		actual_count++;	
	}
	
	fclose(fp);
	
	hipMalloc(&d_a, size_string);
	hipMalloc(&d_b, size_int);
	hipMalloc(&d_c, size_int);
	hipMalloc(&copy_db, size_int);
	hipMalloc(&left, size_int);
	
	hipMemcpy(d_a, strings, size_string, hipMemcpyHostToDevice);
	hipMemcpy(d_b, counts, size_int, hipMemcpyHostToDevice);
	hipMemcpy(d_c, merged, size_int, hipMemcpyHostToDevice);
	//cudaMemcpy(left, any_left, sizeof(int), cudaMemcpyHostToDevice);
	
	int threads_num = 512, blocks_num;
	blocks_num = (int)ceil((float)actual_count/threads_num);
	
	do {
	populate<<<blocks_num, threads_num>>>(d_b, copy_db, d_c, actual_count,left); 	
	select<<<blocks_num, threads_num>>>(copy_db, actual_count);	
	search<<<blocks_num, threads_num>>>(d_b, d_c, actual_count);
	compare<<<blocks_num, threads_num>>>(d_a, d_b, d_c, actual_count, lenString, threshold);
        anyLeft<<<blocks_num, threads_num>>>(d_c, left, actual_count);
	hipMemcpy(any_left, left, sizeof(int), hipMemcpyDeviceToHost);
	} while (*any_left == 0);
		
	hipMemcpy(counts, d_b, size_int, hipMemcpyDeviceToHost);
	hipMemcpy(merged, d_c, size_int, hipMemcpyDeviceToHost);
	
	int counter=0;
	for(int i =0; i<actual_count;i++) {
		if (merged[i] == 2)
			counter++;
	}

	printf("%d\n", counter);
	
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	free(strings);
	free(counts);
	free(merged);
}
