
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>
#include <time.h>
#include <cmath>
#include <cstdlib>
#include <cstring>


int lenString=594;
int maxNumStrings = 1000000;                           
int threshold = 2;

__global__ void populate (int *d_b, int *copy_db, int *d_c, int size) {
	int n = 0;
	int my_id = blockDim.x * blockIdx.x + threadIdx.x;

	if (my_id < size) {
		n = abs((bool)d_c[my_id] - 1);
		copy_db[my_id] = d_b[my_id] * n;
	}		
}
__global__ void cuda_select(int *db, int size, int *largest)
{
	int my_id = blockDim.x * blockIdx.x + threadIdx.x;
	if(my_id < size)
	{
		if(db[2*my_id] > db[2*my_id + 1])
			db[my_id] = db[2*my_id];
		else
			db[my_id] = db[2*my_id + 1];
	}
}
__host__ void select(int *db, int size, int *largest)
{
	int height = (int)ceil(log2((double)size));
	int i = 0;
	
	for(i = 0; i < height; i++)
	{
		size = (int)ceil((double) size/2);
		int threads_num = 512, blocks_num;
		blocks_num = (int)ceil((float)size/threads_num);
		cuda_select<<<blocks_num, threads_num>>>(db, size, largest);
	}
	*largest = db[0];
}
__global__ void Compare(int position, char *d_a, int *d_b, int *d_c, int max_count, int lenString, int threshold){
	int my_id = blockDim.x * blockIdx.x + threadIdx.x;
	if ((my_id < max_count) && (d_c[my_id] == 0) && (my_id != position)){
		int offset = my_id*lenString - position*lenString;
		int x, i, diffs = 0, stop =0;
		for (x=0;x<lenString;x+=6){
			for (i=0;i<6;i++){
				diffs += (bool)(d_a[x+i+position]^d_a[x+i+offset+position]);
				if (diffs > threshold){
					stop += 1;
					break;}
				}
			if (stop == 1)
			break;
	}
	
	if (diffs <= threshold){
		d_b[position] += d_b[my_id];
		d_c[position] = 2;
		d_c[my_id] = 1;
	}
	
	else 
	{
		d_c[position] = 2;
	}
}
}


int main(int argc, char** argv) {//allocation of variables
	char *strings, *d_a;
	int *counts, *merged, *d_b, *d_c;	//host copy of a
	int *largest, *copy_db;
	char copy[lenString+1]; //string to copy in info
	int numbers=0;
	int i=0, actual_count=0;
	int size_string = maxNumStrings*sizeof(char)*(lenString+1);
	int size_int = maxNumStrings*sizeof(int);
	struct timeval start, end; 				//using time
	double wallTime;
	hipError_t status = (hipError_t)0;



	//opening the file
	FILE *fp;
	fp=fopen("/cluster/home/charliep/courses/cs360/single-linkage-clustering/Iceland2014.trim.contigs.good.unique.good.filter.unique.count.fasta", "r");


	if (!(strings= (char *)malloc(size_string))) {
		fprintf(stderr, "malloc() FAILED (Block)\n"); 
		exit(0);}
	if (!(counts= (int*)malloc(size_int))) {
		fprintf(stderr, "malloc() FAILED (Block)\n"); 
		exit(0);}
	merged = (int *)malloc(size_int);	
	


	while( fscanf(fp,"%s %d", copy, &numbers) != EOF && actual_count <100){
		strncpy(&strings[i],copy, lenString);
		counts[actual_count]=numbers;
		//printf("%s\n", copy);
		//printf("%s\n", &a[i]);
		i=i+lenString;
		actual_count++;
		
		}
	fclose(fp);
	
	hipMalloc(&d_a, size_string);
	hipMalloc(&d_b, size_int);
	hipMalloc(&d_c, size_int);
	hipMalloc(&copy_db, size_int);
	
	for(int i =0; i<actual_count;i++)
	{
		printf("%d , %d\n", counts[i], merged[i]);
	}
	
	
	hipMemcpy(d_a, strings, size_string, hipMemcpyHostToDevice);
	hipMemcpy(d_b, counts, size_int, hipMemcpyHostToDevice);
	hipMemcpy(d_c, merged, size_int, hipMemcpyHostToDevice);
	
	int threads_num = 512, blocks_num;
	blocks_num = (int)ceil((float)actual_count/threads_num);
	
	populate<<<blocks_num, threads_num>>>(d_b, copy_db, d_c, actual_count);
	select(copy_db, actual_count, largest);
	
	int position = 0;
	Compare<<<blocks_num, threads_num>>>(position, d_a, d_b, d_c, actual_count, lenString, threshold);
		
	hipMemcpy(counts, d_b, size_int, hipMemcpyDeviceToHost);
	hipMemcpy(merged, d_c, size_int, hipMemcpyDeviceToHost);
	
	printf("\n");
	
	for(int i =0; i<actual_count;i++)
	{
		printf("%d , %d\n", counts[i], merged[i]);
	}
		
		
		
		
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	free(strings);
	free(counts);
	free(merged);
}
