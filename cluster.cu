#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <time.h>
#include <cmath>
#include <cstdlib>
#include <cstring>
#include <hip/hip_runtime.h>

__device__ int position;
int lenString=594;
int maxNumStrings = 1000000;                           
int threshold = 2;

__global__ void kernel() { position = 3;}

__global__ void search(int *d_b, int *d_c, int max_count)
{
	int my_id = blockDim.x * blockIdx.x + threadIdx.x;
	if(d_c[my_id]==0 && (d_b[my_id] > d_b[position])&&(my_id < max_count))
	{
		position = my_id;
	}
}

__global__ void Compare(char *d_a, int *d_b, int *d_c, int max_count, int lenString, int threshold){
	int my_id = blockDim.x * blockIdx.x + threadIdx.x;
	
		
	if ((my_id < max_count) && (d_c[my_id] == 0) && (my_id != position)){
		
		int x, i, diffs = 0, stop =0;
		for (x=0;x<lenString;x++){
			diffs += (bool)(d_a[(lenString*position)+x]^d_a[(my_id*lenString)+x]);
			
			if (diffs > threshold){
				break;}
		}
		
		if (diffs <= threshold){
			d_b[position] += d_b[my_id];
			d_c[position] = 2;
			d_c[my_id] = 1;
			
		}
	
		else {
			d_c[position] = 2;
		}
	}

}


int main(int argc, char** argv) {//allocation of variables
	char *strings, *d_a;
	int *counts, *merged, *d_b, *d_c;	//host copy of a
	char copy[lenString+1]; //string to copy in info
	int numbers=0;
	int i=0, actual_count=0;
	int size_string = maxNumStrings*sizeof(char)*(lenString+1);
	int size_int = maxNumStrings*sizeof(int);
	int size_int_2 = sizeof(int);
	struct timeval start, end; 				//using time
	double wallTime;
	hipError_t status = (hipError_t)0;



	//opening the file
	FILE *fp;
	fp=fopen("/cluster/home/charliep/courses/cs360/single-linkage-clustering/Iceland2014.trim.contigs.good.unique.good.filter.unique.count.fasta", "r");


	if (!(strings= (char *)malloc(size_string))) {
		fprintf(stderr, "malloc() FAILED (Block)\n"); 
		exit(0);}
	if (!(counts= (int*)malloc(size_int))) {
		fprintf(stderr, "malloc() FAILED (Block)\n"); 
		exit(0);}
	merged = (int *)malloc(size_int);	
	hipMemset(&position,0,sizeof(int));

	while( fscanf(fp,"%s %d", copy, &numbers) != EOF && actual_count <100){
		strcpy(&strings[i],copy);
		counts[actual_count]=numbers;
		//printf("%s\n", copy);
		//printf("%s\n", &a[i]);
		i=i+lenString;
		actual_count++;
		
		}
	fclose(fp);
	
	hipMalloc(&d_a, size_string);
	hipMalloc(&d_b, size_int);
	hipMalloc(&d_c, size_int);
	
	for(int i =0; i<actual_count;i++)
	{
		printf("%d , %d\n", counts[i], merged[i]);
	}
	
	
	hipMemcpy(d_a, strings, size_string, hipMemcpyHostToDevice);
	hipMemcpy(d_b, counts, size_int, hipMemcpyHostToDevice);
	hipMemcpy(d_c, merged, size_int, hipMemcpyHostToDevice);
	
	int threads_num = 512, blocks_num;
	blocks_num = (int)ceil((float)actual_count/threads_num);
	
	
	
	kernel<<<1,1>>>();
	//search<<<blocks_num, threads_num>>>(d_b, d_c, actual_count);
	Compare<<<blocks_num, threads_num>>>(d_a, d_b, d_c, actual_count, lenString, threshold);
	

	
	hipMemcpy(counts, d_b, size_int, hipMemcpyDeviceToHost);
	hipMemcpy(merged, d_c, size_int, hipMemcpyDeviceToHost);
	
	printf("\n");
	
	for(int i =0; i<actual_count;i++)
	{
		printf("%d , %d\n", counts[i], merged[i]);
	}
		
		
		
		
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	free(strings);
	free(counts);
	free(merged);
}
