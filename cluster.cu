#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <time.h>
#include <cmath>
#include <cstdlib>
#include <cstring>
#include <hip/hip_runtime.h>

__device__ int position;			//index of the largest value
__device__ int largest;				//value of the largest value
int lenString=594;
int maxNumStrings = 1000000;                           
int threshold = 2;

__global__ void kernel(int set) { position = set;}

__global__ void search(int *d_b, int *d_c, int max_count)
{
	int my_id = blockDim.x * blockIdx.x + threadIdx.x;
	if(d_c[my_id]==0 && (d_b[my_id] == largest )&&(my_id < max_count))
	{
		position = my_id;
	}
}


__global__ void populate (int *d_b, int *copy_db, int *d_c, int size) {
	int n = 0;
	int my_id = blockDim.x * blockIdx.x + threadIdx.x;

	if (my_id < size) {
		n = abs((bool)d_c[my_id] - 1);
		copy_db[my_id] = d_b[my_id] * n;
	}		
}
__device__ void cuda_select(int *db, int size)
{
	int my_id = blockDim.x * blockIdx.x + threadIdx.x;
	if(my_id < size)
	{
		if(db[2*my_id] > db[2*my_id + 1])
			db[my_id] = db[2*my_id];
		else
			db[my_id] = db[2*my_id + 1];
	}
	
}
__global__ void select(int *db, int size)
{
	int height = (int)ceil(log2((double)size));
	int i = 0;
	
	for(i = 0; i < height; i++)
	{
		size = (int)ceil((double) size/2);
		//int threads_num = 512, blocks_num;
		//blocks_num = (int)ceil((float)size/threads_num);
		cuda_select(db, size);
	}
	largest = db[0];
}

__global__ void Compare(char *d_a, int *d_b, int *d_c, int max_count, int lenString, int threshold){

	int my_id = blockDim.x * blockIdx.x + threadIdx.x;
	
		
	if ((my_id < max_count) && (d_c[my_id] == 0) && (my_id != position)){
		
		int x, i, diffs = 0, stop =0;

		for (x=0;x<lenString;x++){
			diffs += (bool)(d_a[(lenString*position)+x]^d_a[(my_id*lenString)+x]);
			
			if (diffs > threshold){
				break;}
		}
		
		if (diffs <= threshold){
			d_b[position] += d_b[my_id];
			d_c[position] = 2;
			d_c[my_id] = 1;
			
		}
		else {
			d_c[position] = 2;
		}
	}

}


int main(int argc, char** argv) {//allocation of variables
	char *strings, *d_a;
	int *counts, *merged, *d_b, *d_c;	//host copy of a
	int *largest, *copy_db, *copy_copy_db;
	char copy[lenString+1]; //string to copy in info
	int numbers=0;
	int i=0, actual_count=0;
	int size_string = maxNumStrings*sizeof(char)*(lenString+1);
	int size_int = maxNumStrings*sizeof(int);
	int size_int_2 = sizeof(int);
	struct timeval start, end; 				//using time
	double wallTime;
	hipError_t status = (hipError_t)0;

	int set_value;


	//opening the file
	FILE *fp;
	fp=fopen("/cluster/home/charliep/courses/cs360/single-linkage-clustering/Iceland2014.trim.contigs.good.unique.good.filter.unique.count.fasta", "r");


	if (!(strings= (char *)malloc(size_string))) {
		fprintf(stderr, "malloc() FAILED (Block)\n"); 
		exit(0);}
	if (!(counts= (int*)malloc(size_int))) {
		fprintf(stderr, "malloc() FAILED (Block)\n"); 
		exit(0);}
	merged = (int *)malloc(size_int);
	copy_copy_db = (int *)malloc(size_int);
	hipMemset(&position,0,sizeof(int));
	hipMemset(&largest,0,sizeof(int));

	while( fscanf(fp,"%s %d", copy, &numbers) != EOF && actual_count <100){
		strcpy(&strings[i],copy);
		counts[actual_count]=numbers;
		//printf("%s\n", copy);
		//printf("%s\n", &a[i]);
		i=i+lenString;
		actual_count++;
		
		}
	fclose(fp);
	
	hipMalloc(&d_a, size_string);
	hipMalloc(&d_b, size_int);
	hipMalloc(&d_c, size_int);
	hipMalloc(&copy_db, size_int);
	
	for(int i =0; i<actual_count;i++)
	{
		printf("%d , %d\n", counts[i], merged[i]);
	}
	
	
	hipMemcpy(d_a, strings, size_string, hipMemcpyHostToDevice);
	hipMemcpy(d_b, counts, size_int, hipMemcpyHostToDevice);
	hipMemcpy(d_c, merged, size_int, hipMemcpyHostToDevice);
	
	int threads_num = 512, blocks_num;
	blocks_num = (int)ceil((float)actual_count/threads_num);


	
	populate<<<blocks_num, threads_num>>>(d_b, copy_db, d_c, actual_count); //this does what it is suppose to do so far checking next part
	select<<<blocks_num, threads_num>>>(copy_db, actual_count);							//not selecting the largest value
	
	kernel<<<1,1>>>(set_value);
	search<<<blocks_num, threads_num>>>(d_b, d_c, actual_count);
	Compare<<<blocks_num, threads_num>>>(d_a, d_b, d_c, actual_count, lenString, threshold);

		
	hipMemcpy(counts, d_b, size_int, hipMemcpyDeviceToHost);
	hipMemcpy(merged, d_c, size_int, hipMemcpyDeviceToHost);
	
	printf("\n");
	
	
	for(int i =0; i<actual_count;i++)
	{
		printf("%d , %d\n", counts[i], merged[i]);
	}
	
		
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	free(strings);
	free(counts);
	free(merged);
}
